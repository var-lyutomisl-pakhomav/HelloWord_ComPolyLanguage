
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA()
{
    printf("Hello, CUDA world from thread %d!\n", threadIdx.x);
}

int main()
{
    helloCUDA<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
